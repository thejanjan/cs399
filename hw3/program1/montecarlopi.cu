#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

const int THREADS = 16;

__global__ void monte_carlo(int *point_counts) {
    // some consts for this function
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // setup cuRAND
    hiprandState state;
    hiprand_init((unsigned long long)clock(), idx, 0, &state);

    // calculate success for this thread
    float x = hiprand_uniform(&state);
    float y = hiprand_uniform(&state);
    point_counts[idx] = 0;
    if (((x * x) + (y * y)) < 1.0)
        point_counts[idx] = 1;
}

__global__ void reduce(int *gdata, int *out, int N) {
	// grid-strided reduction code from lecture 5
	__shared__ float sdata[THREADS];
	int tid = threadIdx.x;
	sdata[tid] = 0;
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	while (idx < N) {
		sdata[tid] += gdata[idx];
		idx += gridDim.x*blockDim.x;
	}
	for (unsigned int s=blockDim.x/2;s>0;s>>=1) {
		__syncthreads();
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
	}
	if (tid == 0) {
		atomicAdd(out, sdata[0]);
	}
}

int main(int argc, char *argv[]) {
    // Handle option inputs -- really rough arg parsing
    int num_points = 1000;
    if (argc == 3) {
        if (strcmp(argv[1], "-numpoints") == 0) {
            num_points = atoi(argv[2]);
        }
    }

    ///
    /// Monte Carlo algorithm
    ///
	
	printf("Running with %d points\n", num_points);

    // allocate host and device memory
    int BLOCKS = (num_points + THREADS - 1);

    int *point_counts, *d_point_counts;
	int *out, *d_out;
    point_counts = (int *)malloc(num_points * sizeof(int));
	out = (int *)malloc(sizeof(int));
    hipMalloc(&d_point_counts, num_points * sizeof(int));
	hipMalloc(&d_out, sizeof(int));
	printf("Memory allocated\n");

    // perform kernel
    monte_carlo<<<BLOCKS, THREADS>>>(d_point_counts);
	reduce<<<BLOCKS, THREADS>>>(d_point_counts, d_out, num_points);
	printf("Kernel performed\n");

    // collect result
    hipMemcpy(out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_point_counts);
	hipFree(d_out);
	printf("Memory freed\n");

    // calculate pi, display results
    float pi_approx = 4.0f * ((float)(*out) / (float)num_points);
    printf("Number of points: %d\n", num_points);
    printf("Points within quarter circle: %d\n", *out);
    printf("Pi approximate: %f\n", pi_approx);

    // cleanup and return
    free(point_counts);
	free(out);
    return 0;
}
