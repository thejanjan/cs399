#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const int THREADS = 32;

///
/// benchmark functions
///

float _bm_startTime;

void start_benchmark() {
	_bm_startTime = (float)clock()/CLOCKS_PER_SEC;
}

void end_benchmark(char *name) {
	float _bm_endTime = (float)clock()/CLOCKS_PER_SEC;
	float timeElapsed = _bm_endTime - _bm_startTime;
	printf("-=- %s -=-\n", name);
	printf("Time elapsed: %f ms\n", timeElapsed * 1000.0f);
}

///
/// kernels
///

__global__ void matrix_init(float *a, float *a2, float *b, float *b2, float *c, int m, int n, float value) {
	int xi = threadIdx.x + blockIdx.x*blockDim.x;
	int yi = threadIdx.y + blockIdx.y*blockDim.y;
	
	int i = xi + (yi * n);
	
	if (i < (n * n)) {
		c[i] = value;
		
		if (i < (n * m)) {
			a[i] = value;
			b2[i] = value;
			a2[i] = value;
			b[i] = value;
		}
	}
}

__global__ void matrix_mult(float *A, float *B, float *C, int m, int n) {
	// calculate the x and y index the thread is working on
	int xi = threadIdx.x + blockIdx.x*blockDim.x;
	int yi = threadIdx.y + blockIdx.y*blockDim.y;
	
	int i = yi + (xi * n);

	if (xi < n && yi < n) {
		// perform sum
		float result = 0.0f;
		for (int k = 0; k < m; k++) {
			int ai = k + (xi * n);  // A[i][k]
			int bi = yi + (k * n);  // B[k][j]
			result += A[ai] * B[bi];
		}

		// set output
		C[i] = result;
	}
}

__global__ void matrix_mult_b2(float *A, float *B2, float *C, int m, int n) {
	// calculate the x and y index the thread is working on
	int xi = threadIdx.x + blockIdx.x*blockDim.x;
	int yi = threadIdx.y + blockIdx.y*blockDim.y;

	int i = yi + (xi * n);

	if (xi < n && yi < n) {
		// perform sum
		float result = 0.0f;
		for (int k = 0; k < m; k++) {
			int ai = k + (xi * n);
			int bi = k + (xi * n);
			result += A[ai] * B2[bi];
		}

		// set output
		C[i] = result;
	}
}

__global__ void matrix_mult_a2(float *A2, float *B, float *C, int m, int n) {
	// calculate the x and y index the thread is working on
	int xi = threadIdx.x + blockIdx.x*blockDim.x;
	int yi = threadIdx.y + blockIdx.y*blockDim.y;

	int i = yi + (xi * n);

	if (xi < n && yi < n) {
		// perform sum
		float result = 0.0f;
		for (int k = 0; k < m; k++) {
			int ai = yi + (k * n);
			int bi = yi + (k * n);
			result += A2[ai] * B[bi];
		}

		// set output
		C[i] = result;
	}
}

///
/// main
///

int main(int argc, char *argv[]) {
	// Handle option inputs -- really rough arg parsing
	int n = 100;
	int m = 100;

	if (argc == 5) {
		if (strcmp(argv[1], "-n") == 0) {
			n = atoi(argv[2]);
		}
		if (strcmp(argv[3], "-m") == 0) {
			m = atoi(argv[4]);
		}
	}

	///
	/// Mandelbrot calculations
	///

	// setup cuda
	int max_dim = max(n, m);
	dim3 threads_per_block(THREADS, THREADS);
	dim3 blocks_per_grid(ceil((float)max_dim/(float)THREADS), ceil((float)max_dim/(float)THREADS));

	float *a_d, *a2_d, *b_d, *b2_d, *c_d, *c;
	c = (float *)malloc(n * n * sizeof(float));
	hipMalloc(&a_d,  n * m * sizeof(float));
	hipMalloc(&a2_d, n * m * sizeof(float));
	hipMalloc(&b_d,  n * m * sizeof(float));
	hipMalloc(&b2_d, n * m * sizeof(float));
	hipMalloc(&c_d,  n * n * sizeof(float));

	// init matrices
	matrix_init<<<blocks_per_grid, threads_per_block>>>(a_d, a2_d, b_d, b2_d, c_d, m, n, 1.0f);

	// do initial computation, benchmark
	start_benchmark();
	matrix_mult<<<blocks_per_grid, threads_per_block>>>(a_d, b_d, c_d, m, n);
	end_benchmark("A*B");

	// check results (sorry doc... not doing this with cuda because i'm a REBELLIOUS TEEN!!!)
	hipMemcpy(c, c_d, sizeof(float) * n * n, hipMemcpyDeviceToHost);
	int fails = 0;
	for (int i = 0; i < (n * n); i++) {
		if (c[i] != m) {
			printf("Verify error at %d, val is %f (should be %d)\n", i, c[i], m);
			fails++;
		}
	}
	if (fails > 0) {
		printf("Verify misses: %d\n", fails);
	}

	// transpose B, benchmark
	start_benchmark();
	matrix_mult_b2<<<blocks_per_grid, threads_per_block>>>(a_d, b2_d, c_d, m, n);
	end_benchmark("A*B, transposed B");

	// transpose A, benchmark
	start_benchmark();
	matrix_mult_a2<<<blocks_per_grid, threads_per_block>>>(a2_d, b_d, c_d, m, n);
	end_benchmark("A*B, transposed A");

	// return and cleanup
	free(c);
	hipFree(a_d);
	hipFree(a2_d);
	hipFree(b_d);
	hipFree(b2_d);
	hipFree(c_d);
	return 0;
}
